
#include <hip/hip_runtime.h>
#include <cassert>

#ifdef _WIN32
using uint = unsigned int;
using uchar = unsigned char;
using ushort = unsigned short;
using int64_t = long long;
using uint64_t = unsigned long long;
#else
#define uint unsigned int
#define uchar unsigned char
#define ushort unsigned short
#define int64_t long long
#define uint64_t unsigned long long
#endif

extern "C" __global__ void __launch_bounds__(64)
    bs1_stage3_v(float *__restrict__ v, float *__restrict__ raw) {
    v[((((int) blockIdx.x) * 64) + ((int) threadIdx.x))] =
        (((((((int) blockIdx.x) * 13) + ((int) threadIdx.x)) % 17) == 16)
             ? 1.000000e+00f
             : raw[(
                   ((((((int) blockIdx.x) / 4352) * 786432) +
                     ((((((int) blockIdx.x) * 13) + ((int) threadIdx.x)) % 17) *
                      16384)) +
                    ((((((int) blockIdx.x) % 4352) * 64) +
                      ((int) threadIdx.x)) /
                     17)) +
                   524288)]);
}

extern "C" __global__ void __launch_bounds__(64)
    bs1_stage3_q(float *__restrict__ q, float *__restrict__ raw) {
    q[((((int) blockIdx.x) * 64) + ((int) threadIdx.x))] =
        ((0.000000e+00f < raw[(((((((int) blockIdx.x) >> 12) * 786432) +
                                 ((((int) threadIdx.x) & 15) * 16384)) +
                                ((((int) blockIdx.x) & 4095) * 4)) +
                               (((int) threadIdx.x) >> 4))])
             ? raw[(((((((int) blockIdx.x) >> 12) * 786432) +
                      ((((int) threadIdx.x) & 15) * 16384)) +
                     ((((int) blockIdx.x) & 4095) * 4)) +
                    (((int) threadIdx.x) >> 4))]
             : 0.000000e+00f);
}

extern "C" __global__ void __launch_bounds__(64)
    bs1_stage3_k(float *__restrict__ k, float *__restrict__ raw) {
    k[((((int) blockIdx.x) * 64) + ((int) threadIdx.x))] =
        ((0.000000e+00f < raw[(((((((int) blockIdx.x) >> 12) * 786432) +
                                 ((((int) blockIdx.x) & 4095) * 64)) +
                                ((int) threadIdx.x)) +
                               262144)])
             ? raw[(((((((int) blockIdx.x) >> 12) * 786432) +
                      ((((int) blockIdx.x) & 4095) * 64)) +
                     ((int) threadIdx.x)) +
                    262144)]
             : 0.000000e+00f);
}

extern "C" __global__ void __launch_bounds__(64)
    bs1_stage4_v(float *__restrict__ v, float *__restrict__ raw) {
    v[((((int) blockIdx.x) * 64) + ((int) threadIdx.x))] =
        (((((((int) blockIdx.x) * 13) + ((int) threadIdx.x)) % 17) == 16)
             ? 1.000000e+00f
             : raw[(
                   ((((((int) blockIdx.x) / 1088) * 196608) +
                     ((((((int) blockIdx.x) * 13) + ((int) threadIdx.x)) % 17) *
                      4096)) +
                    ((((((int) blockIdx.x) % 1088) * 64) +
                      ((int) threadIdx.x)) /
                     17)) +
                   131072)]);
}

extern "C" __global__ void __launch_bounds__(64)
    bs1_stage4_q(float *__restrict__ q, float *__restrict__ raw) {
    q[((((int) blockIdx.x) * 64) + ((int) threadIdx.x))] =
        ((0.000000e+00f < raw[(((((((int) blockIdx.x) >> 10) * 196608) +
                                 ((((int) threadIdx.x) & 15) * 4096)) +
                                ((((int) blockIdx.x) & 1023) * 4)) +
                               (((int) threadIdx.x) >> 4))])
             ? raw[(((((((int) blockIdx.x) >> 10) * 196608) +
                      ((((int) threadIdx.x) & 15) * 4096)) +
                     ((((int) blockIdx.x) & 1023) * 4)) +
                    (((int) threadIdx.x) >> 4))]
             : 0.000000e+00f);
}

extern "C" __global__ void __launch_bounds__(64)
    bs1_stage4_k(float *__restrict__ k, float *__restrict__ raw) {
    k[((((int) blockIdx.x) * 64) + ((int) threadIdx.x))] =
        ((0.000000e+00f < raw[(((((((int) blockIdx.x) >> 10) * 196608) +
                                 ((((int) blockIdx.x) & 1023) * 64)) +
                                ((int) threadIdx.x)) +
                               65536)])
             ? raw[(((((((int) blockIdx.x) >> 10) * 196608) +
                      ((((int) blockIdx.x) & 1023) * 64)) +
                     ((int) threadIdx.x)) +
                    65536)]
             : 0.000000e+00f);
}

void membound(
    int bs, int feat_len, int w, float *raw, float *q, float *k, float *v) {
    if (w == 96) {
        bs1_stage3_q<<<49152, 64>>>(q, raw);
        bs1_stage3_k<<<49152, 64>>>(k, raw);
        bs1_stage3_v<<<52224, 64>>>(v, raw);
    } else {
        bs1_stage4_q<<<24576, 64>>>(q, raw);
        bs1_stage4_k<<<24576, 64>>>(k, raw);
        bs1_stage4_v<<<26112, 64>>>(v, raw);
    }
}